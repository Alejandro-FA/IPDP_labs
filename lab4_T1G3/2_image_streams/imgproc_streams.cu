/*
 *     
 *  IMAGE PROCESSING
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define pixel(i, j, n)  (((j)*(n)) +(i))


/*read*/
void  readimg(char * filename,int nx, int ny, int * image){
  
   FILE *fp=NULL;

   fp = fopen(filename,"r");
   for(int j=0; j<ny; ++j){
      for(int i=0; i<nx; ++i){
         fscanf(fp,"%d", &image[pixel(i,j,nx)]);      
      }
   }
   fclose(fp);
}

/* save */   
void saveimg(char *filename,int nx,int ny,int *image){

   FILE *fp=NULL;
   fp = fopen(filename,"w");
   for(int j=0; j<ny; ++j){
      for(int i=0; i<nx; ++i){
         fprintf(fp,"%d ", image[pixel(i,j,nx)]);      
      }
      fprintf(fp,"\n");
   }
   fclose(fp);

}

/*invert*/
__global__ void invert(int* image, int* image_invert, int nx, int ny){

}

/*smooth*/
__global__ void smooth(int* image, int* image_smooth, int nx, int ny){

}

/*detect*/
__global__ void detect(int* image, int* image_detect, int nx, int ny){
   
}

/*enhance*/
__global__ void enhance(int* image,int *image_enhance,int nx, int ny){
   

}

/* Main program */
int main (int argc, char *argv[])
{
   int    nx,ny;
   char   filename[250];

   /* Get parameters */
   if (argc != 4) 
   {
      printf ("Usage: %s image_name N M \n", argv[0]);
      exit (1);
   }
   sprintf(filename, "%s.txt", argv[1]);
   nx  = atoi(argv[2]);
   ny  = atoi(argv[3]);

   printf("%s %d %d\n", filename, nx, ny);

   /* Allocate CPU and GPU pointers */

   int*   image=(int *) malloc(sizeof(int)*nx*ny); 
   int*   image_invert  = (int *) malloc(sizeof(int)*nx*ny);  
   int*   image_smooth  = (int *) malloc(sizeof(int)*nx*ny);  
   int*   image_detect  = (int *) malloc(sizeof(int)*nx*ny);  
   int*   image_enhance = (int *) malloc(sizeof(int)*nx*ny); 
  

   /* Read image and save in array imgage */
   readimg(filename,nx,ny,image);


  /* Print runtime */

   
   /* Save images */
   char fileout[255]={0};
   sprintf(fileout, "%s-inverse.txt", argv[1]);
   saveimg(fileout,nx,ny,image_invert);
   sprintf(fileout, "%s-smooth.txt", argv[1]);
   saveimg(fileout,nx,ny,image_smooth);
   sprintf(fileout, "%s-detect.txt", argv[1]);
   saveimg(fileout,nx,ny,image_detect);
   sprintf(fileout, "%s-enhance.txt", argv[1]);
   saveimg(fileout,nx,ny,image_enhance);

   /* Deallocate CPU and GPU pointers*/
   free(image);
   free(image_invert);
   free(image_smooth);
   free(image_detect);
   free(image_enhance);
}
