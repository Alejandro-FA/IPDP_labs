/*
 *     
 *  IMAGE PROCESSING
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"

#define pixel(i, j, n)  (((j)*(n)) +(i))


/*read*/
void  readimg(char * filename,int nx, int ny, int * image){
  
   FILE *fp=NULL;

   fp = fopen(filename,"r");
   for(int j=0; j<ny; ++j){
      for(int i=0; i<nx; ++i){
         fscanf(fp,"%d", &image[pixel(i,j,nx)]);      
      }
   }
   fclose(fp);
}

/* save */   
void saveimg(char *filename,int nx,int ny,int *image){

   FILE *fp=NULL;
   fp = fopen(filename,"w");
   for(int j=0; j<ny; ++j){
      for(int i=0; i<nx; ++i){
         fprintf(fp,"%d ", image[pixel(i,j,nx)]);      
      }
      fprintf(fp,"\n");
   }
   fclose(fp);
}

// invert
__global__ void invert(int* image, int* image_invert, int nx, int ny) {
   int i = threadIdx.x + blockIdx.x * blockDim.x;
   int j = threadIdx.y + blockIdx.y * blockDim.y;

   if (i < nx && j < ny) {
      int p = pixel(i,j,nx);
      image_invert[p] = 255 - image[p];
   }
}


// smooth
__global__ void smooth(int* image, int* image_smooth, int nx, int ny){
   int i = threadIdx.x + blockIdx.x * blockDim.x;
   int j = threadIdx.y + blockIdx.y * blockDim.y;

   if (i < nx && j < ny) {
      if (i == 0 || i == nx-1 || j == 0 || j == ny-1) image_smooth[pixel(i,j,nx)] = 0;
      else {
         image_smooth[pixel(i,j,nx)] = (
            image[pixel(i-1,j+1,nx)] + 
            image[pixel(i,j+1,nx)] +
            image[pixel(i+1,j+1,nx)] +
            image[pixel(i-1,j,nx)] +
            image[pixel(i,j,nx)] +
            image[pixel(i+1,j,nx)]+
            image[pixel(i-1,j-1,nx)] +
            image[pixel(i,j-1,nx)] +
            image[pixel(i+1,j-1,nx)] ) / 9;

         // Ensure that the pixel value is between 0 and 255 
         image_smooth[pixel(i,j,nx)] = image_smooth[pixel(i,j,nx)] < 0 ? 0 : image_smooth[pixel(i,j,nx)] > 255 ? 255 : image_smooth[pixel(i,j,nx)];
      }
   }
}

// detect
__global__ void detect(int* image, int* image_detect, int nx, int ny){
   int i = threadIdx.x + blockIdx.x * blockDim.x;
   int j = threadIdx.y + blockIdx.y * blockDim.y;
   
   if (i < nx && j < ny) {
      if (i == 0 || i == nx-1 || j == 0 || j == ny-1) image_detect[pixel(i,j,nx)] = 0;
      else {
         image_detect[pixel(i,j,nx)] =
            image[pixel(i-1,j,nx)] +
            image[pixel(i+1,j,nx)] +
            image[pixel(i,j-1,nx)] +
            image[pixel(i,j+1,nx)] -
            4 * image[pixel(i,j,nx)];

         // Ensure that the pixel value is between 0 and 255 
         image_detect[pixel(i,j,nx)] = image_detect[pixel(i,j,nx)] < 0 ? 0 : image_detect[pixel(i,j,nx)] > 255 ? 255 : image_detect[pixel(i,j,nx)];
      }
   }
}

// enhance
__global__ void enhance(int* image,int *image_enhance,int nx, int ny){
   int i = threadIdx.x + blockIdx.x * blockDim.x;
   int j = threadIdx.y + blockIdx.y * blockDim.y;
   
   if (i < nx && j < ny) {
      if (i == 0 || i == nx-1 || j == 0 || j == ny-1) image_enhance[pixel(i,j,nx)] = 0;
      else {
         image_enhance[pixel(i,j,nx)] = 5 * image[pixel(i,j,nx)] - (
            image[pixel(i-1,j,nx)] +
            image[pixel(i+1,j,nx)] +
            image[pixel(i,j-1,nx)] +
            image[pixel(i,j+1,nx)] );

         // Ensure that the pixel value is between 0 and 255 
         image_enhance[pixel(i,j,nx)] = image_enhance[pixel(i,j,nx)] < 0 ? 0 : image_enhance[pixel(i,j,nx)] > 255 ? 255 : image_enhance[pixel(i,j,nx)];  
      }
   }
}

/* Main program */
int main (int argc, char *argv[])
{
   int    nx,ny;
   char   filename[250];

   /* Get parameters */
   if (argc != 4) 
   {
      printf ("Usage: %s image_name N M \n", argv[0]);
      exit (1);
   }
   sprintf(filename, "%s.txt", argv[1]);
   nx  = atoi(argv[2]);
   ny  = atoi(argv[3]);

   printf("%s %d %d\n", filename, nx, ny);

   /* Allocate CPU pointers */
   int   *h_image, *d_image; 
   int   *h_image_invert, *d_image_invert;
   int   *h_image_smooth, *d_image_smooth;
   int   *h_image_detect, *d_image_detect;
   int   *h_image_enhance, *d_image_enhance;

   hipHostMalloc((void**)&h_image, sizeof(int)*nx*ny, hipHostMallocDefault);
   hipHostMalloc((void**)&h_image_invert, sizeof(int)*nx*ny, hipHostMallocDefault);
   hipHostMalloc((void**)&h_image_smooth, sizeof(int)*nx*ny, hipHostMallocDefault);
   hipHostMalloc((void**)&h_image_detect, sizeof(int)*nx*ny, hipHostMallocDefault);
   hipHostMalloc((void**)&h_image_enhance, sizeof(int)*nx*ny, hipHostMallocDefault);
   
   /* Read image and save in array imgage */
   readimg(filename,nx,ny,h_image);

   /* Decide Block and Grid dimensions */
   int B = 16; // Number of blocks (maximum number of blocks that can be simultaneously active)
   dim3 dimBlock(B, B, 1);
   int dimgx = (nx+B-1)/B;
   int dimgy = (ny+B-1)/B;
   dim3 dimGrid(dimgx, dimgy, 1);

   float runtime;
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   /************************************ Start recording ************************************/
   hipEventRecord(start);

   /* Create Streams */
   hipStream_t stream1, stream2, stream3, stream4;
   hipStreamCreate(&stream1);
   hipStreamCreate(&stream2);
   hipStreamCreate(&stream3);
   hipStreamCreate(&stream4);

   /* Allocate GPU pointers */
   hipHostMalloc((void**)&d_image, sizeof(int)*nx*ny, hipHostMallocDefault);
   hipHostMalloc((void**)&d_image_invert, sizeof(int)*nx*ny, hipHostMallocDefault);
   hipHostMalloc((void**)&d_image_smooth, sizeof(int)*nx*ny, hipHostMallocDefault);
   hipHostMalloc((void**)&d_image_detect, sizeof(int)*nx*ny, hipHostMallocDefault);
   hipHostMalloc((void**)&d_image_enhance, sizeof(int)*nx*ny, hipHostMallocDefault);

   /* Copy image to GPU */
   hipMemcpy(d_image, h_image, sizeof(int)*nx*ny, hipMemcpyHostToDevice);  // Sync version since we need it in all streams

   /* Filters */
   invert<<<dimGrid, dimBlock, 0, stream1>>>(d_image, d_image_invert, nx, ny);
   smooth<<<dimGrid, dimBlock, 0, stream2>>>(d_image, d_image_smooth, nx, ny);
   detect<<<dimGrid, dimBlock, 0, stream3>>>(d_image, d_image_detect, nx, ny);
   enhance<<<dimGrid, dimBlock, 0, stream4>>>(d_image, d_image_enhance, nx, ny);
   
   /* Image transfer */
   hipMemcpyAsync(h_image_invert, d_image_invert, sizeof(int)*nx*ny, hipMemcpyDeviceToHost, stream1); 
   hipMemcpyAsync(h_image_smooth, d_image_smooth, sizeof(int)*nx*ny, hipMemcpyDeviceToHost, stream2);
   hipMemcpyAsync(h_image_detect, d_image_detect, sizeof(int)*nx*ny, hipMemcpyDeviceToHost, stream3);
   hipMemcpyAsync(h_image_enhance, d_image_enhance, sizeof(int)*nx*ny, hipMemcpyDeviceToHost, stream4);
   
   /* Delete streams */
   hipStreamDestroy(stream1);
   hipStreamDestroy(stream2);
   hipStreamDestroy(stream3);
   hipStreamDestroy(stream4);

   hipEventRecord(stop);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&runtime, start, stop);
   /************************************* End recording *************************************/

   printf("It took %f ms to apply all the filters and to manage all the data. \n", runtime);

   /* Save images */
   char fileout[255]={0};
   sprintf(fileout, "%s-inverse.txt", argv[1]);
   saveimg(fileout,nx,ny,h_image_invert);
   sprintf(fileout, "%s-smooth.txt", argv[1]);
   saveimg(fileout,nx,ny,h_image_smooth);
   sprintf(fileout, "%s-detect.txt", argv[1]);
   saveimg(fileout,nx,ny,h_image_detect);
   sprintf(fileout, "%s-enhance.txt", argv[1]);
   saveimg(fileout,nx,ny,h_image_enhance);

   /* Deallocate GPU pointers */
   hipFree(d_image);
   hipFree(d_image_invert);
   hipFree(d_image_smooth);
   hipFree(d_image_detect);
   hipFree(d_image_enhance);

   /* Deallocate CPU pointers*/
   hipHostFree(h_image);
   hipHostFree(h_image_invert);
   hipHostFree(h_image_smooth);
   hipHostFree(h_image_detect);
   hipHostFree(h_image_enhance);
}
